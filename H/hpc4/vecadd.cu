
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void add(int*A , int*B , int*C , int size)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if(tid<size)
    {
        C[tid] = A[tid]+B[tid];
    }
    
}

void initialize(int *vector , int size)
{
    for(int i = 0 ; i<size ; i++)
    {
        vector[i] = rand()%10;
    }
}

void print(int *vector , int size)
{
    for(int i= 0 ; i<size ; i++)
    {
        cout<<vector[i]<<" ";
    }
}

int main()
{
    int N = 10;
    int vectorsize = N;

    size_t vectorbytes = vectorsize*sizeof(int);
    int *A ,*B,*C;

    A = new int[vectorsize];   //allocate
    B = new int[vectorsize];
    C = new int[vectorsize];

    initialize(A,N);
    initialize(B,N);

    cout<<"A : ";
    print(A,N);
    cout<<endl;

    cout<<"B : ";
    print(B,N);
    cout<<endl;

    int *X,*Y,*Z;

    hipMalloc(&X, vectorbytes);  //memory allocate
    hipMalloc(&Y, vectorbytes);
    hipMalloc(&Z, vectorbytes);

    hipMemcpy(X , A , vectorbytes , hipMemcpyHostToDevice);  //memory allocate from local to cuda
    hipMemcpy(Y , B , vectorbytes , hipMemcpyHostToDevice);

    int threadsperblock = 256;       
    int blockspergrid = (N+ threadsperblock -1)/threadsperblock;   //to see if all elements are convered 

    add<<<blockspergrid, threadsperblock>>>(X,Y,Z,N);

    hipMemcpy(C, Z , vectorbytes , hipMemcpyDeviceToHost);

    cout<<"C : ";
    print(C,N);
    cout<<endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    





    

}