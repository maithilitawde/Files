
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__ void multiply (int *A , int *B , int *C, int size)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if(row<size && col<size)
    {
        int sum = 0 ;
        for(int i = 0 ; i<size ; i++)
        {
            sum += A[row*size + i]*B[i*size + col];
        }
        C[row*size+col] = sum;
    }
}

void initialize(int *matrix , int size)
{
    for(int i = 0 ;i<size ; i++)
    {
        matrix[i] = rand()%10;
    }
}

void print(int *matrix , int size)
{
    for(int row = 0 ; row<size ; row++)
    {
        for(int col = 0 ; col<size ; col++)
        {
            cout<<matrix[row*size+col]<<" ";
        }
        cout<<endl;
    }
}

int main()
{
    int N = 2;
    int matrixsize = N*N;
    size_t matrixbytes = matrixsize*sizeof(int);

    int *A, *B, *C;

    A = new int[matrixsize];
    B = new int[matrixsize];
    C = new int[matrixsize];

    initialize(A,matrixsize);
    initialize(B,matrixsize);

    cout<<"A : "<<endl;
    print(A,N);
    cout<<endl;

    cout<<"B : "<<endl;
    print(B,N);
    cout<<endl;

    int *X,*Y,*Z;

    hipMalloc(&X , matrixbytes);
    hipMalloc(&Y , matrixbytes);
    hipMalloc(&Z , matrixbytes);

    hipMemcpy(X,A,matrixbytes , hipMemcpyHostToDevice);
    hipMemcpy(Y,B,matrixbytes , hipMemcpyHostToDevice);

    int Threads = 2;
    int Blocks = (N)/Threads;

    dim3 threads(Threads, Threads);
    dim3 blocks(Blocks,Blocks);

    multiply<<<blocks,threads>>>(X,Y,Z,N);

    hipMemcpy(C,Z,matrixbytes , hipMemcpyDeviceToHost);
    cout<<"C : "<<endl;
    print(C,N);
    cout<<endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);







}